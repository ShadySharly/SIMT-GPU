
#include <hip/hip_runtime.h>
# include <getopt.h>
# include <ctype.h>
# include <stdlib.h>
# include <stdio.h>
# include <unistd.h>
# include <string.h>
# include <pmmintrin.h>
# include <time.h>

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void suma2D_CPU(float* A, float* B, int N, int V);

void getParams (int argc, char** argv, char* nValue, char* bValue, char* vValue);

int isInteger (char* input);

float pixelSum (float* image, int N);

void printImage (float* image, int N);

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void suma2D (float* A, float* B, int N, int V) {
    
    int offset, neighbour, mid_row, neigh_row, center_neigh;

    int local_i = threadIdx.x;
    int local_j = threadIdx.y;

    int global_i = blockDim.x * blockIdx.x + local_i;
    int global_j = blockDim.y * blockIdx.y + local_j;

    int global_id = global_i + global_j * N;

    B[global_id] = 0.0;

    for (offset = -V * (1 + N); offset <= V * (1 + N); offset++) {
        neighbour = global_id + offset;
        neigh_row = neighbour / N;
        mid_row = global_id / N;
        
        // Condicion para no considerar vecinos fuera de los limites de la imagen
        if ( (neighbour >= 0) && (neighbour < (N * N)) ) {
            center_neigh = global_id - (mid_row - neigh_row) * N;

            // Condicion para no considerar vecinos fuera de la vecindad
            if ( (neighbour >= (center_neigh - V)) && (neighbour <= (center_neigh + V)) ) {
                B[global_id] = B[global_id] + A[neighbour];
            }
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__host__ int main(int argc, char** argv) {
    
    clock_t start_t, end_t;
    float sum_gpu, sum_seq, gpu_time, cpu_time;
    char* nValue = (char*)malloc(sizeof(char));
    char* bValue = (char*)malloc(sizeof(char)); 
    char* vValue = (char*)malloc(sizeof(char));

    getParams (argc, argv, nValue, bValue, vValue);

    int N = atoi(nValue);
    int Bs = atoi(bValue);
    int V = atoi(vValue);

    dim3 gridSize = dim3(N / Bs, N / Bs);
    dim3 blockSize = dim3(Bs, Bs);
    
    float* h_a = (float*)malloc( (N * N) * sizeof(float));
    float* h_b = (float*)malloc( (N * N) * sizeof(float));
    float* seq_b = (float*)malloc( (N * N) * sizeof(float));

    float* d_a;
    float* d_b;

    // SE LLENA LA IMAGEN CON VALORES ALEATORIOS
    for (int index = 0; index < (N * N); index++) {
        h_a[index] = (float) rand() / RAND_MAX; 
    }

    // Se crean y se inicializan los eventos para capturar el tiempo de ejecucion para 
    // todas las operaciones que se realizan utilizando la GPU.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void**) &d_a, (N * N) * sizeof(float));
    hipMalloc((void**) &d_b, (N * N) * sizeof(float));

    hipMemcpy(d_a, h_a, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, (N * N) * sizeof(float), hipMemcpyHostToDevice);

    suma2D<<<gridSize, blockSize>>>(d_a, d_b, N, V);

    hipMemcpy(h_b, d_b, (N * N) * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    sum_gpu = pixelSum (h_b, N);
    
    printf("Tiempo GPU: %f (ms)\n", gpu_time);
    printf("Suma GPU: %f\n", sum_gpu);

    start_t = clock();

    suma2D_CPU (h_a, seq_b, N, V);

    end_t = clock();
    cpu_time = (float)(end_t - start_t) / CLOCKS_PER_SEC;
    cpu_time *= 1000;

    sum_seq = pixelSum (seq_b, N);

    printf("Tiempo CPU: %f (ms)\n", cpu_time);
    printf("Suma CPU: %f\n", sum_seq);

    printImage(h_a, N);
    printf("\n");
    printImage(h_b, N);

    // Destruccion de los eventos iniciados
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Liberacion de memoria para el host y el device.
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    // Liberacion de memoria para la recepcion de parametros de entrada.
    free(nValue);
    free(bValue);
    free(vValue);

    return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void suma2D_CPU(float* A, float* B, int N, int V) {

    int index, offset, neighbour, mid_row, neigh_row, center_neigh;

    for (index = 0; index < (N * N); index++){
        B[index] = 0.0;

        for (offset = -V * (1 + N); offset <= V * (1 + N); offset++) {
            neighbour = index + offset;
            neigh_row = neighbour / N;
            mid_row = index / N;
            
            // Condicion para no considerar vecinos fuera de los limites de la imagen
            if ( (neighbour >= 0) && (neighbour < (N * N)) ) {
                center_neigh = index - (mid_row - neigh_row) * N;
    
                // Condicion para no considerar vecinos fuera de la vecindad
                if ( (neighbour >= (center_neigh - V)) && (neighbour <= (center_neigh + V)) ) {
                    B[index] = B[index] + A[neighbour];
                }
            }
        } 
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// - INPUTS: - argc: Largo del arreglo de argumentos argv.
//           - argv: Arreglo con los argumentos de entrada incluyendo en nombre del archivo.
//           - iValue: Nombre del archivo de entrada que contiene el archivo en formato binario (RAW).
//           - oValue: Nombre del archivo de salida con las secuencia ordenada en formato binario (RAW).
//           - nValue: Largo de la secuencia contenida en el archivo de entrada (Numero entero multiplo de 16).
//           - dValue: Bandera que controla el debug para imprimir los resultados por consola (1) o no (0).
// - OUTPUTS: -
// - DESCRIPTION: Procedimiento que obtiene los parametros entregados por consola y almacenados en la variable "argv", y los deposita en las variables
//                iValue, oValue, nValue y dValue, en cada caso verificando la validez del valor entragado para cada bandera. Si alguna de estas banderas
//                no cumple con los formatos especificados el programa es interrumpido.

void getParams (int argc, char** argv, char* nValue, char* bValue, char* vValue) {

    int c;
    while ( (c = getopt (argc, argv, "N:B:V:")) != -1) {

        switch (c) {
            case 'N':
                strcpy(nValue, optarg);
                if (!isInteger(nValue)) {
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    printf (" => El argumento de -%c debe ser un ENTERO POSITIVO.\n", c);
                    printf (" => Programa abortado\n");
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    exit(EXIT_FAILURE);
                }

                break;

            case 'B':
                strcpy(bValue, optarg);
                if (!isInteger(bValue)) {
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    printf (" => El argumento de -%c debe ser un ENTERO POSITIVO.\n", c);
                    printf (" => Programa abortado\n");
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    exit(EXIT_FAILURE);
                }

                break;
            
            case 'V':
                strcpy(vValue, optarg);
                if (!isInteger(vValue)) {
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    printf (" => El argumento de -%c debe ser un ENTERO POSITIVO.\n", c);
                    printf (" => Programa abortado\n");
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    exit(EXIT_FAILURE);
                }

                break;

            case '?':
                if ( (optopt == 'N') || (optopt == 'B') || (optopt == 'V') ) { 
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    printf (" => La opcion -%c requiere un argumento.\n", optopt);
                    printf (" => Programa abortado\n");
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    exit(EXIT_FAILURE);
                }

                else if (isprint (optopt)) {
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    printf (" => Opcion -%c desconocida.\n", optopt);
                    printf (" => Programa abortado\n");
                    printf ("%s\n", "-------------------------------------------------------------------------");
                    exit(EXIT_FAILURE);
                }

            default:
                break;
            }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// - INPUTS: - input: Cadena de caracteres a evaluar si corresponde a un numero entero positivo o no
// - OUTPUTS: Valor booleano 1 si es entero positivo, 0 en caso contrario
// - DESCRIPTION: Verifica si una cadena de caracteres de entrada posee en cada una de sus posiciones un caracter que es
//                digito y es positivo

int isInteger (char* input) {

    int c;
    // Recorrer el argumento entragado en cadena de caracteres, verificando que cada uno de estos corresponde a un numero.
    for (c = 0; c < strlen(input); c++) {

        // Si no se cumple para alguno de los caracteres, significa que el argumento no corresponde a un entero positivo y retorna 0.
        if (!isdigit(input[c]))
            return 0;
    }
    return 1;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// - INPUTS: - input: Cadena de caracteres a evaluar si corresponde a un numero entero positivo o no
// - OUTPUTS: Valor booleano 1 si es entero positivo, 0 en caso contrario
// - DESCRIPTION: Verifica si una cadena de caracteres de entrada posee en cada una de sus posiciones un caracter que es

float pixelSum (float* image, int N) {

    int index;
    float sum = 0.0;

    for (index = 0; index < (N * N); index++) {
        sum += image[index];
    }

    return sum;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// - INPUTS: - input: Cadena de caracteres a evaluar si corresponde a un numero entero positivo o no
// - OUTPUTS: Valor booleano 1 si es entero positivo, 0 en caso contrario
// - DESCRIPTION: Verifica si una cadena de caracteres de entrada posee en cada una de sus posiciones un caracter que es

void printImage (float* image, int N) {

    for (int index = 0; index < (N * N); index++) {
        printf("%f ", image[index]);

        if ((index + 1) % N == 0)
            printf("\n");
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////// END ////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////